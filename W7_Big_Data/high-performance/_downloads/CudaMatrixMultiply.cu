/*
nvcc -o foo.out
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// global declarations
typedef struct {
  int width;
  int height;
  float* elements;
} Matrix;

// functions
int print_mat_by_row(const Matrix M){
  int i,j;
  int n,d;
  
  n = M.height;
  d = M.width;
  
  printf("\nprinting matrix rows -- size: %d, rank: %d\n",n,d);
  for (i=0; i<n; i++)
    {
      printf("row(%d)[",i);
      for (j=0; j<d; j++)
  	{
	  printf(" %g ",M.elements[i * M.width + j]);
	}
      printf("]\n");
    }
  printf("\n");
  return 0;
}

__global__ void matrix_multiply_kernel(Matrix A, const Matrix B, const Matrix C, int aHeight, int aWidth, int bHeight, int bWidth){
  int x,j,k;
  float p;
  x = blockIdx.x;
  
  if (x < aHeight){    
    for(j=0; j< bWidth; j++){       // the num of cols in B
      p=0;                          // reset product to 0
      for(k=0; k < aWidth; k++){    // the num of cols in  A awa num rows in B
	p+=A.elements[x * aWidth + k] * B.elements[k * bWidth + j];
      }
      C.elements[x * bWidth + j] = p;
    }
  }
}

int main (void)
{
  // allocate host variables
  Matrix A,B,C;
  A.height = 4;
  A.width = 2;
  A.elements = (float*) malloc(A.width * A.height * sizeof(float));

  B.height = 2;
  B.width = 3;
  B.elements = (float*) malloc(B.width * B.height * sizeof(float));

  C.height = A.height;
  C.width = B.width;
  C.elements = (float*) malloc(C.width * C.height * sizeof(float));

  // allocate device variables
  Matrix dev_A, dev_B, dev_C;
  hipMalloc((void**) &dev_A.elements, A.height * A.width * sizeof(float));
  hipMalloc((void**) &dev_B.elements, B.height * B.width * sizeof(float));
  hipMalloc((void**) &dev_C.elements, C.height * C.width * sizeof(float));

  // populate host matrices -- in general M.elements[row * M.width + col] = value; 
  A.elements[0 * A.width + 0] = 1;
  A.elements[0 * A.width + 1] = 2;
  A.elements[1 * A.width + 0] = 3;
  A.elements[1 * A.width + 1] = 4;
  A.elements[2 * A.width + 0] = 5;
  A.elements[2 * A.width + 1] = 6;
  A.elements[3 * A.width + 0] = 7;
  A.elements[3 * A.width + 1] = 8;

  B.elements[0 * B.width + 0] = 1;
  B.elements[0 * B.width + 1] = 2;
  B.elements[0 * B.width + 2] = 3;
  B.elements[1 * B.width + 0] = 4;
  B.elements[1 * B.width + 1] = 5;
  B.elements[1 * B.width + 2] = 6;

  // print out the matrices
  print_mat_by_row(A);
  print_mat_by_row(B);
   
  // copy to device
  hipMemcpy(dev_A.elements, A.elements, A.height * A.width * sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(dev_B.elements, B.elements, B.height * B.width * sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(dev_C.elements, C.elements, C.height * C.width * sizeof(float),hipMemcpyHostToDevice);

  // invoke the kernel
  matrix_multiply_kernel<<<A.height,1>>>(dev_A,dev_B,dev_C, A.height, A.width, B.height, B.width);

  // Read C from device memory
  hipMemcpy(C.elements, dev_C.elements, C.height * C.width * sizeof(float),hipMemcpyDeviceToHost);
  print_mat_by_row(C);
    
  // free up memory
  hipFree(dev_A.elements);
  hipFree(dev_B.elements);
  hipFree(dev_C.elements);

  free(A.elements);
  free(B.elements);
  free(C.elements);
  
  return 0;
}
